#include "hip/hip_runtime.h"
/*
 * William Setzer
 * wjsetzer
 * CS 732
 * Homework 1
 * */

#include <stdbool.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include <time.h>

#include "game.cuh"

// void copy_board(board_t, board_t);
// bool arraycomp(uint64_t*, uint64_t*, uint64_t);
__device__
void check_neighbors(bool*, const uint64_t, const uint64_t);

bool* new_board(const uint64_t height, const uint64_t width) {
  /*
   * Return a 1-dimensional array (N+2)x(M+2) long with random values.
   * The values are initialized from 1..N and 1..M.
   * The edges are treated as ghost cells.
   * */

  // First and last N+2 are false.
  // For every chunk, the first and last are false.
  //
  // allocate contiguous memory
  bool* cells = (bool*) calloc((height + 2) * (width + 2), sizeof(bool));
  
  return cells; 

}

bool* random_board(const uint64_t height, const uint64_t width, const double chance, const uint64_t seed) {
  /*
   * Return a 1-dimensional array (N+2)x(M+2) long with random values.
   * The values are initialized from 1..N and 1..M.
   * The edges are treated as ghost cells.
   * */

  // First and last N+2 are false.
  // For every chunk, the first and last are false.
  
  // srand((uint32_t)time(NULL));
  srand(seed);

  bool* board = new_board(height, width);

  size_t y = 1;
  for (y = 1; y <= height; y++) {
    size_t x = 1;
    for (x = 1; x <= width; x++) {
      int randomnum = rand();
      bool val = chance > ((double) randomnum / (double) RAND_MAX);
      if (val) {
        board[y * height + x] = true;
        // not necessarily true, but ensure the cell will be
        // examined in the first iteration
        // count_neighbors(board, y, x);
      }
    }
  }


  return board; 
}

/*
cyclesum_t newcyclesum(const uint64_t row, const uint64_t col, uint64_t maxcycles) {

  maxcycles = maxcycles + 1; // add 1 to make space for the cycle being summed

  // create an array for holding sums of length maxcycle
  uint64_t** const sums = calloc(sizeof(uint64_t*), maxcycles);

  size_t x = 0;
  for (x = 0; x < maxcycles; x++) {

    // create an array for holding row sums of length col
    sums[x] = calloc(sizeof(uint64_t), col);

  }

  cyclesum_t cyclesum = {
    maxcycles,
    col,
    row,
    0, // current cycle is currently 0
    sums
  };

  return cyclesum;
}
*/

/*
void freecyclesum(cyclesum_t cyclesum) {
  size_t x = 0;
  for (x = 0; x < cyclesum.maxcycle; x++) {
    free(cyclesum.sums[x]);
  }

  free(cyclesum.sums);
}

bool arraycomp(uint64_t* a, uint64_t* b, uint64_t len) {

  size_t x = 0;
  for (x = 0; x < len; x++) {
    if (a[x] != b[x]) return false;
  }

  return true;
}
*/

/*
bool checkcycles(cyclesum_t* cyclesum, board_t board) {
   Returns true if two cycles are equal, else false
   * Sets the current state of the board to be the next
   * cycle, replacing the oldest cycle
   * 

  // sum the alive cells in the rows of the board
  
  size_t y = 1;
  for (y = 1; y <= board.height; y++ ) {
    uint64_t sum = 0;

    size_t x = 1;
    for (x = 1; x <= board.width; x++) {
      if (board.cells[y * board.height + x]) {
        sum++;
      }
    }

    // set sum to cyclesum current
    cyclesum->sums[cyclesum->current][y - 1] = sum;
  }

  // compare current to other cycles
  for (y = 0; y < cyclesum->maxcycle; y++) {
    uint64_t currentcycle = cyclesum->current;

    if (y != currentcycle) {
      if ( arraycomp(cyclesum->sums[y], cyclesum->sums[currentcycle], cyclesum->row)) {
        return true;
      }
    }
  }

  // increment cycle
  // NOTE: there is no need to increment if two cycles matched
  cyclesum->current = (cyclesum->current + 1) % cyclesum->maxcycle;

  return false;
}
*/


__device__
uint_fast8_t check_neighbors(const bool* board, const uint64_t height, const uint64_t width, const uint64_t y_pos, const uint64_t x_pos) {
  /*
   * increments the neighbors field of the surrounding neighbor
   * cells.
   * */

  uint_fast8_t count = 0;

  int y = -1;
  for (y = -1; y <= 1; y++) {
    int x = -1;
    for (x = -1; x <= 1; x++) {

      if (x == 0 && y == 0) {
        continue;
      }

      if (board[(y + y_pos) * (height + 2) + x + x_pos + 2]) {
        count++;
      }
    }
  }

  return count;

}

__global__
void advance_board(const bool* const current_gen, bool* const next_gen, const uint64_t height, const uint64_t width) {


  // uint64_t x = blockDim.x * blockIdx.x + threadIdx.x;
  // uint64_t y = blockDim.y * blockIdx.y + threadIdx.y;

  uint64_t index = blockIdx.x * blockDim.x + threadIdx.x;
  // uint64_t yindex = blockIdx.y * blockDim.y + threadIdx.y;

  // index will be between 1 and height * width

  uint64_t y = index / (height + 2); 

  uint64_t x = index - (y * (height + 2));

  if (y < 1 || y > height || x < 1 || x > width) {
    // if out of bounds
    return;
  }

  bool current_cell = current_gen[y * (height + 2) + x];

  uint_fast8_t count = check_neighbors(current_gen, height, width, y, x);

  if (current_cell && ( count > 3 || count < 2)) {
    current_cell = false;
  }
  else if (! current_cell && count == 3) {
    current_cell = true;
  }

  next_gen[y * (height + 2) + x + 2] = current_cell;
}

__global__
void white_board(bool* const board, const uint64_t height, const uint64_t width) {

  uint64_t xindex = blockIdx.x * blockDim.x + threadIdx.x;
  uint64_t yindex = blockIdx.y * blockDim.y + threadIdx.y;

  uint64_t xstride = blockDim.x;
  uint64_t ystride = blockDim.y;

  for (size_t y = yindex; y <= height; y += ystride) {
    for (size_t x = xindex; x <= width; x += xstride) {

      board[y * height + x] = true;
    }
  }
}

/*
void copy_board(board_t dest, board_t src) {
  size_t y = 0;
  for (y = 0; y < src.height + 2; y++) {
    memcpy(dest.cells[y], src.cells[y], sizeof(bool) * src.height + 2);
  }
}
*/

void printboard(bool* board, const uint64_t height, const uint64_t width) {
  size_t y = 1;
  for (y = 1; y <= height; y++ ){
    size_t x = 1;
    for (x = 1; x <= width; x++) {
      printf(board[y * height + x] ? "█" : "░");
    }
    puts("");
  }
}

void printfullboard(bool* board, const uint64_t height, const uint64_t width) {
  size_t y = 0;
  for (y = 0; y < height + 2; y++ ){
    size_t x = 0;
    for (x = 0; x < width + 2; x++) {
      printf(board[y * height + x] ? "█" : "░");
    }
    puts("");
  }
}

/*
void free_board(board_t board) {
  free(board.cells);
}
*/

