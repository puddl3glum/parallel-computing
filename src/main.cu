
/*
 * William Setzer
 * wjsetzer
 * CS 732
 * Homework 1
 * */

#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <unistd.h>

#include "main.cuh"
#include "game.cuh"

// int getopt(int argc, char * const argv[], const char *optstring);
// extern char* optarg;
// extern int optind, opterr, optopt;
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
  if (code != hipSuccess){
    fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);

    if (abort) {
      exit(code);
    }
  }
}

int main(int argc, char* argv[]) {


  int opt;

  uint64_t width = 100;
  uint64_t height = 100;
  uint64_t generations = 100;
  double chance = 0.5;
  uint64_t seed = 1;


  while ((opt = getopt(argc, argv, "s:w:h:g:c:")) != -1) {

    switch(opt) {
      case 's':
        // set the seed
        seed = strtoul(optarg, NULL, 10);
        break;
      case 'w':
        width = strtoul(optarg, NULL, 10);
        break;
      case 'h':
        height = strtoul(optarg, NULL, 10);
        break;
      case 'g':
        generations = strtoul(optarg, NULL, 10);
        break;
      case 'c':
        chance = strtod(optarg, NULL);
        break;
      default:
        printf("USAGE: %s [OPTION]\n", argv[0]);
        puts("Options:");
        puts("\ts : The seed to use for the random number generator.");
        puts("\tw : The width for the board.");
        puts("\th : The height for the board.");
        puts("\tg : The number of generations for the game.");
        puts("\tc : The chance a cell will be alive.");
        return 0;
    }
  }

  // Read in board state
  // OR
  // Randomize board state
  bool* current_gen = random_board(height, width, chance, seed);
  bool* next_gen = new_board(height, width);

  // memcpy(next_gen, current_gen, (height + 2) * (width + 2) * sizeof(bool));
  // get new cyclesum tracker
  // cyclesum_t cyclesum = newcyclesum(width, height, maxcycles);
  
  // create CUDA space
  bool* cuda_current_gen;
  bool* cuda_next_gen;
  gpuErrchk(hipMalloc(&cuda_current_gen, (height + 2) * (width + 2) * sizeof(bool)));
  gpuErrchk(hipMalloc(&cuda_next_gen, (height + 2) * (width + 2) * sizeof(bool)));

  // printf("%p\n", cuda_current_gen);
  // printf("%p\n", cuda_next_gen);

  // memcpy(cuda_current_gen, current_gen, (height + 2) * (width + 2) * sizeof(bool));
  // puts("here");

  hipMemcpy(cuda_current_gen, current_gen, (height + 2) * (width + 2) * sizeof(bool), hipMemcpyHostToDevice);
  // hipMemcpy(cuda_next_gen, current_gen, (height + 2) * (width + 2) * sizeof(bool), hipMemcpyHostToDevice);

  current_gen = new_board(height, width);

  // hipMemcpy(current_gen, cuda_current_gen, (height + 2) * (width + 2) * sizeof(bool), hipMemcpyDeviceToHost);
  printboard(cuda_current_gen, height, width);

  uint64_t blocksize = 512;
  uint64_t numblocks =  ((height) * (width) + blocksize - 1) / blocksize;

  size_t gen = 0;
  for (gen = 0; gen < generations; gen++) {

    // Simulate generation
    advance_board<<<numblocks, blocksize>>>(cuda_current_gen, cuda_next_gen, height, width);

    hipDeviceSynchronize();
    bool* temp;

    temp = cuda_current_gen;
    cuda_current_gen = cuda_next_gen;
    cuda_next_gen = temp;

    // if ( checkcycles(&cyclesum, board) ) break;

    // compute the checksum of the board
    // sum alive cells in each row
    // add to array
    // default, 1 generation
    // if current array == any in history : quit
    
    // break;
  }

  white_board<<<numblocks, blocksize>>>(cuda_next_gen, height, width);

  // hipMemcpy(current_gen, cuda_current_gen, (height + 2) * (width + 2), hipMemcpyDeviceToHost);
  hipMemcpy(next_gen, cuda_next_gen, (height + 2) * (width + 2) * sizeof(bool), hipMemcpyDeviceToHost);

#ifdef DEBUG
  printboard(current_gen, height, width);
  puts("");
  printboard(next_gen, height, width);
#endif
  
  // cleanup
  hipFree(cuda_current_gen);
  hipFree(cuda_next_gen);
  free(current_gen);
  free(next_gen);
  // freecyclesum(cyclesum);
}

